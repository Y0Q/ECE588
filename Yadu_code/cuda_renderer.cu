#include "hip/hip_runtime.h"
//test
#include <algorithm>
#include <math.h>
#include <stdio.h>
#include <vector>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <driver_functions.h>

#include "cuda_renderer.h"
#include "image.h"

struct globals_const {

    SceneName sceneName;

    int 	numCircles;
    float* 	position;
    float* 	velocity;
    float* 	color;
    float* 	radius;

    int 	imgWidth;
    int 	imgHeight;
    float* 	imgData;	
};

//constants for GPU to access
__constant__ globals_const cuConstParams;

//Clearing the image to initial snowflakes setting
__global__ void kernelClearImageSnowflake(){
	
	int image_X = blockIdx.x * blockDim.x + threadIdx.x;
	int image_Y = blockIdx.y * blockDim.y + threadIdx.y;

	int width 	= cuConstParams.imgWidth;
	int height 	= cuConstParams.imgHeight;
	
	if(image_X >= width || image_Y >= height){
		return;
	}
	
	int 	offset 	= 4* (image_Y * width + image_X);
	float	shader 	= 0.4f + 0.45f * static_cast<float>(height - image_Y) / height;
	float4 	value  	= make_float4(shader,shader,shader,1.f);
	
	//Writing it to GPU memory
	*(float4*)(&cuConstParams.imgData[offset]) = value;
}

Cuda_renderer::Cuda_renderer() {
    image = NULL;

    numCircles = 0;
    position = NULL;
    velocity = NULL;
    color = NULL;
    radius = NULL;

    cudaDevicePosition = NULL;
    cudaDeviceVelocity = NULL;
    cudaDeviceColor = NULL;
    cudaDeviceRadius = NULL;
    cudaDeviceImageData = NULL;
}

//Allocating buffer memory to the image.
void Cuda_renderer::allocImageBuf(int width, int height{
		
		if(image){
			delete image;
		}
		image = new Image(width,height);
)

//Loading the scene
void Cuda_renderer::loadScene(SceneName scene){
	sceneName = scene;
	
	if(sceneName == SNOWFLAKES){
		//Write an algorithm
	}
}

static void genRandomCircle(  int 		numCircles,
							  float*	position,
							  float*	velocity,
							  float*	color,
							  float*	radius){

		srand(0);
		std::vector<float> depths()
								  
}

//Clearing image for the renderer
void Cuda_renderer::clearImage(){
	
	//256 threads per blockDim
	dim3 blockDim(16,16,1);
	dim3 gridDim( 
				(image->width  + blockDim.x - 1) / blockDim.x,
				(image->height + blockDim.y - 1) / blockDim.y
	);
	
	if(sceneName == SNOWFLAKES){
		kernelClearImageSnowflake<<gridDim, blockDim>>();
	}else{
		//KernelClearImage call
	}
	hipDeviceSynchronize();
}





















